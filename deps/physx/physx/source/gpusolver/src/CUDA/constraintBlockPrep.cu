#include "hip/hip_runtime.h"
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ''AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Copyright (c) 2008-2025 NVIDIA Corporation. All rights reserved.
// Copyright (c) 2004-2008 AGEIA Technologies, Inc. All rights reserved.
// Copyright (c) 2001-2004 NovodeX AG. All rights reserved.

#include "PxgBodySim.h"
#include "PxgArticulation.h"
#include "PxgSolverBody.h"
#include "PxgConstraint.h"
#include "PxgFrictionPatch.h"
#include "PxgConstraintPrep.h"
#include "PxgSolverConstraintDesc.h"
#include "PxgSolverCoreDesc.h"
#include "PxgCudaMemoryAllocator.h"
#include "PxgArticulationCoreKernelIndices.h"
#include "DySolverConstraintTypes.h"
#include "DyConstraintPrep.h"
#include "PxNodeIndex.h"
#include "PxContact.h"
#include "PxsContactManagerState.h"
#include "contactConstraintBlockPrep.cuh"
#include "contactConstraintPrep.cuh"
#include "jointConstraintBlockPrep.cuh"
#include "constant.cuh"
#include "constraintPrepShared.cuh"
#include <assert.h>
#include "stdio.h"


using namespace physx;

extern "C" __host__ void initSolverKernels1() {}

#define LOAD_BODY_DATA 0

#if LOAD_BODY_DATA

//Enough memory to fit 32 warps and load 11 solver body data objects per-pass, i.e. load solverBodyData for all 32 warps in 3 passes.
//Note, we can +1 on the size to avoid bank conflicts but then 16 byte aligned structs won't be aligned anymore
#define BODIES_PER_BLOCK 11u
volatile __shared__ PxU8 bodyLoadData[PxgKernelBlockDim::CONSTRAINT_PREPARE_BLOCK_PARALLEL/32][BODIES_PER_BLOCK][sizeof(PxgSolverBodyPrepData)];


static __device__ void loadBodyData(const PxgSolverBodyData* PX_RESTRICT datas, const PxU32 batchStride, const PxU32 bodyIndex, const PxU32 threadIndexInWarp, const PxU32 warpIndex,
	PxgSolverBodyPrepData& outBodyPrepData/*float4& initialLinVelXYZ_invMassW, float4& initialAngVelXYZ_penBiasClamp, PxAlignedMat33&	sqrtInvInertia, PxAlignedTransform& body2World*/)
{
	//Iterate through the body datas, pulling in the data we need, then index into shared data, pull out the solver body data and return it by value to store on stack (either in register or in local mem).
	threadCounts[threadIdx.x] = bodyIndex;

	const PxU32 solverPrepDataWords = sizeof(PxgSolverBodyPrepData)/4;

	PxU32 warpStartIndex = warpIndex*32;

	for(PxU32 a = 0; a < batchStride; a+=BODIES_PER_BLOCK)
	{
		PxU32 remainder = PxMin(batchStride - a, BODIES_PER_BLOCK);

		for(PxU32 b = 0; b < remainder; ++b)
		{
			PxU32 bodyIndex = threadCounts[warpStartIndex + a + b]; //KS - potentially can use SM3.0 shuffle instead

			const PxU32* PX_RESTRICT sourceData = reinterpret_cast<const PxU32*>(datas + bodyIndex);

			volatile PxU32* bodyData = reinterpret_cast<volatile PxU32*>(&bodyLoadData[warpIndex][b][0]);

			for(PxU32 i = threadIndexInWarp; i < solverPrepDataWords; i+=32)
			{
				bodyData[i] = sourceData[i];
			}

		}

		if((threadIndexInWarp - a) < BODIES_PER_BLOCK)
		{
			volatile PxgSolverBodyPrepData& data = reinterpret_cast<volatile PxgSolverBodyPrepData&>(bodyLoadData[warpIndex][threadIndexInWarp-a][0]);

			/*initialLinVelXYZ_invMassW = make_float4(data.initialLinVelXYZ_invMassW.x, data.initialLinVelXYZ_invMassW.y, data.initialLinVelXYZ_invMassW.z,
				initialLinVelXYZ_invMassW.w);
			initialAngVelXYZ_penBiasClamp = make_float4(data.initialAngVelXYZ_penBiasClamp.x, data.initialAngVelXYZ_penBiasClamp.y, data.initialAngVelXYZ_penBiasClamp.z,
				data.initialAngVelXYZ_penBiasClamp.w);

			body2World.p = make_float4(data.body2World.p.x, data.body2World.p.y, data.body2World.p.z, data.body2World.p.w);
			body2World.q = make_float4(data.body2World.q.q.x, data.body2World.q.q.y, data.body2World.q.q.z, data.body2World.q.q.w);

			sqrtInvInertia = (PxAlignedMat33&)data.sqrtInvInertia;*/
			outBodyPrepData = (PxgSolverBodyPrepData&)data;
			
			/*PxU32* outPrepDataU32 = reinterpret_cast<PxU32*>(&outPrepData);
			for(PxU32 i = 0; i < solverPrepDataWords; ++i)
			{
				outPrepDataU32[i] = bodyLoadData[warpIndex][threadIndexInWarp - a][i];
			}*/
		}

	}

	threadCounts[threadIdx.x] = 0; //Reset thread counts to 0 because they're used for accumulators in later code

}

#endif

extern "C" __global__ void jointConstraintBlockPrepareParallelLaunch(
	PxgConstraintPrepareDesc* solverDesc,
	PxgSolverSharedDesc<IterativeSolveData>* sharedDesc)
{
	//threadCounts[threadIdx.x] = 0;

	//__syncthreads();

	//PxgBlockWorkUnit* workUnits = constraintPrepDesc->workUnit;

	const PxU32 warpSize = 32;

	const PxU32 blockStride = blockDim.x/warpSize;

	//This identifies which warp a specific thread is in, we treat all warps in all blocks as a flatten warp array
	//and we are going to index the work based on that
	const PxU32 warpIndex = blockIdx.x * blockStride + threadIdx.x/warpSize;

	//This identifies which thread within a warp a specific thread is
	const PxU32 threadIndexInWarp = threadIdx.x&(warpSize-1);

	//total numbers of warps in all blocks
	//const PxU32 totalNumWarps = blockStride * gridDim.x;

	//PxF32* baseForceStream = constraintPrepDesc->forceBuffer;

	PxgSolverBodyData* solverBodyDatas = solverDesc->solverBodyDataPool;
	PxgSolverTxIData* solverTxIData = solverDesc->solverBodyTxIDataPool;

	PxgBlockSolverConstraint1DHeader* jointConstraintHeaders = sharedDesc->iterativeData.blockJointConstraintHeaders;
	PxgBlockSolverConstraint1DCon* jointConstraintRowsCon = sharedDesc->iterativeData.blockJointConstraintRowsCon;
	PxgBlockSolverConstraint1DMod* jointConstraintRowsMod = sharedDesc->iterativeData.blockJointConstraintRowsMod;
	PxU32* batchIndices = solverDesc->jointConstraintBatchIndices;

	const PxU32 num1dConstraintBatches = solverDesc->num1dConstraintBatches + solverDesc->numStatic1dConstraintBatches;

	//for(PxU32 i=warpIndex; i< constraintPrepDesc->num1dConstraintBatches; i+=totalNumWarps)
	PxU32 i = warpIndex;
	if (i < num1dConstraintBatches)
	{
		const PxU32 batchIndex = batchIndices[i];
		PxgBlockConstraintBatch& batch = sharedDesc->iterativeData.blockConstraintBatch[batchIndex];
		const PxU32 bodyAIndex = batch.bodyAIndex[threadIndexInWarp];
		const PxU32 bodyBIndex = batch.bodyBIndex[threadIndexInWarp];
			
		const PxU32 descIndexBatch = batch.mConstraintBatchIndex;

		const PxU32 descStride = batch.mDescStride;

		//PxgSolverBodyPrepData bodyData0, bodyData1;

#if LOAD_BODY_DATA
		loadBodyData(solverBodyDatas, descStride, bodyAIndex, threadIndexInWarp, warpIndexInBlock, bodyData0.initialLinVelXYZ_invMassW, bodyData0.initialAngVelXYZ_penBiasClamp,
			bodyData0.sqrtInvInertia, bodyData0.body2World);
		loadBodyData(solverBodyDatas, descStride, bodyBIndex, threadIndexInWarp, warpIndexInBlock, bodyData1.initialLinVelXYZ_invMassW, bodyData1.initialAngVelXYZ_penBiasClamp,
			bodyData1.sqrtInvInertia, bodyData1.body2World);
#endif

		//mDescStride might less than 32, we need to guard against it
		if(threadIndexInWarp < descStride)
		{
				//desc.descIndex for joint in fact is the batch index
			PxgBlockConstraint1DData& constraintData = solverDesc->blockJointPrepPool[descIndexBatch];
			PxgBlockConstraint1DVelocities* rowVelocities = &solverDesc->blockJointPrepPool0[descIndexBatch * Dy::MAX_CONSTRAINT_ROWS];
			PxgBlockConstraint1DParameters* rowParameters = &solverDesc->blockJointPrepPool1[descIndexBatch * Dy::MAX_CONSTRAINT_ROWS];

			PxgSolverBodyData* bodyData0 = &solverBodyDatas[bodyAIndex];
			PxgSolverBodyData* bodyData1 = &solverBodyDatas[bodyBIndex];
			PxgSolverTxIData* txIData0 = &solverTxIData[bodyAIndex];
			PxgSolverTxIData* txIData1 = &solverTxIData[bodyBIndex];

			PxU32 uniqueIndex = solverDesc->constraintUniqueIndices[batch.mStartPartitionIndex + threadIndexInWarp];
				
			setupSolverConstraintBlockGPU<PxgKernelBlockDim::CONSTRAINT_PREPARE_BLOCK_PARALLEL>(&constraintData, rowVelocities, rowParameters, bodyData0, bodyData1, txIData0, txIData1, sharedDesc->dt, sharedDesc->invDtF32, batch, threadIndexInWarp,
					&jointConstraintHeaders[descIndexBatch], &jointConstraintRowsCon[batch.startConstraintIndex], &jointConstraintRowsMod[batch.startConstraintIndex],
					solverDesc->solverConstantData[uniqueIndex]);
		}    
	}
}

extern "C" __global__ void contactConstraintBlockPrepareParallelLaunch(
	PxgConstraintPrepareDesc* constraintPrepDesc,
	PxgSolverSharedDesc<IterativeSolveData>* sharedDesc)
{
	//threadCounts[threadIdx.x] = 0;

	//__syncthreads();

	PxgBlockWorkUnit* workUnits = constraintPrepDesc->blockWorkUnit;

	const PxU32 warpSize = WARP_SIZE;

	const PxU32 blockStride = blockDim.x/warpSize;

	//This identifies which warp a specific thread is in, we treat all warps in all blocks as a flatten warp array
	//and we are going to index the work based on that
	const PxU32 warpIndex = blockIdx.x * blockStride + threadIdx.x/warpSize;

	//This identifies which thread within a warp a specific thread is
	const PxU32 threadIndexInWarp = threadIdx.x&(warpSize-1);

	//total numbers of warps in all blocks
	//const PxU32 totalNumWarps = blockStride * gridDim.x;

	//PxF32* baseForceStream = constraintPrepDesc->forceBuffer;

	const PxU32 totalPreviousEdges = constraintPrepDesc->totalPreviousEdges;
	const PxU32 totalCurrentEdges = constraintPrepDesc->totalCurrentEdges;
	const PxU32 nbContactBatches = constraintPrepDesc->numContactBatches + constraintPrepDesc->numStaticContactBatches;


	/*if (warpIndex == 0 && threadIndexInWarp == 0)
	{
		printf("NumBatches = %i, numContactBatches = %i, numStaticContactBatches = %i %p\n", nbContactBatches,
			constraintPrepDesc->numContactBatches, constraintPrepDesc->numStaticContactBatches, constraintPrepDesc);
	}*/

	__shared__ PxgSolverBodyData* solverBodyDatas;
	__shared__ PxgSolverTxIData* solverTxIDatas;

	__shared__ PxgBlockSolverContactHeader* contactHeaders;
	__shared__ PxgBlockSolverFrictionHeader* frictionHeaders;
	__shared__ PxgBlockSolverContactPoint* contactPoints;
	__shared__ PxgBlockSolverContactFriction* frictions;
	__shared__ PxU32* batchIndices;
	__shared__ PxgBlockFrictionIndex* frictionIndices;
	__shared__ PxgBlockFrictionIndex* prevFrictionIndices;
	__shared__ PxgBlockContactPoint* contactBase;
	__shared__ PxgBlockConstraintBatch* constraintBatch;
	__shared__ PxgBlockContactData* contactCurrentPrepPool;
	__shared__ PxgBlockFrictionPatch* prevFrictionPatches;
	__shared__ PxgBlockFrictionPatch* currFrictionPatches;
	__shared__ PxgBlockFrictionAnchorPatch* prevFrictionAnchors;
	__shared__ PxgBlockFrictionAnchorPatch* currFrictionAnchors;
	__shared__ PxAlignedTransform* bodyFrames;

	
	volatile __shared__ char sInertias[sizeof(PxMat33) * (PxgKernelBlockDim::CONSTRAINT_PREPARE_BLOCK_PARALLEL / warpSize) * warpSize];
	//volatile __shared__ PxMat33 inertias[PxgKernelBlockDim::CONSTRAINT_PREPARE_BLOCK_PARALLEL / warpSize][warpSize];

	volatile PxMat33* inertias = reinterpret_cast<volatile PxMat33*>(sInertias);

	if(threadIdx.x == 0)
	{
		solverBodyDatas = constraintPrepDesc->solverBodyDataPool;
		solverTxIDatas = constraintPrepDesc->solverBodyTxIDataPool;

		contactHeaders = sharedDesc->iterativeData.blockContactHeaders;
		frictionHeaders = sharedDesc->iterativeData.blockFrictionHeaders;
		contactPoints = sharedDesc->iterativeData.blockContactPoints;
		frictions = sharedDesc->iterativeData.blockFrictions;
		batchIndices = constraintPrepDesc->contactConstraintBatchIndices;
		frictionIndices = constraintPrepDesc->blockCurrentFrictionIndices;
		prevFrictionIndices = constraintPrepDesc->blockPreviousFrictionIndices;

		contactBase = constraintPrepDesc->blockContactPoints;
		constraintBatch = sharedDesc->iterativeData.blockConstraintBatch;
		contactCurrentPrepPool = constraintPrepDesc->blockContactCurrentPrepPool;
		currFrictionPatches = sharedDesc->blockCurrentFrictionPatches;
		prevFrictionPatches = sharedDesc->blockPreviousFrictionPatches;
		prevFrictionAnchors = constraintPrepDesc->blockPreviousAnchorPatches;
		currFrictionAnchors = constraintPrepDesc->blockCurrentAnchorPatches;
		bodyFrames = constraintPrepDesc->body2WorldPool;
	}
	
	__syncthreads();

	PxU32 i = warpIndex;
	//unsigned mask_nbContactBatches = __ballot_sync(FULL_MASK, i < nbContactBatches);
	if(i < nbContactBatches)
	{
		const PxU32 batchIndex = batchIndices[i];

		//if (batchIndex >= totalBatches)
		//{
		//	if(batchIndices[i-1] < totalBatches)
		//		assert(batchIndex < totalBatches); //Ensure we are not shooting past the max number of batches...
		//}

		PxgBlockConstraintBatch& batch = constraintBatch[batchIndex];
		const PxU32 bodyAIndex = batch.bodyAIndex[threadIndexInWarp];
		const PxU32 bodyBIndex = batch.bodyBIndex[threadIndexInWarp];
			
		const PxU32 descIndexBatch = batch.mConstraintBatchIndex;

		const PxU32 descStride = batch.mDescStride;

		//PxgSolverBodyPrepData bodyData0, bodyData1;

#if LOAD_BODY_DATA
		loadBodyData(solverBodyDatas, descStride, bodyAIndex, threadIndexInWarp, warpIndexInBlock, bodyData0.initialLinVelXYZ_invMassW, bodyData0.initialAngVelXYZ_penBiasClamp,
			bodyData0.sqrtInvInertia, bodyData0.body2World);
		loadBodyData(solverBodyDatas, descStride, bodyBIndex, threadIndexInWarp, warpIndexInBlock, bodyData1.initialLinVelXYZ_invMassW, bodyData1.initialAngVelXYZ_penBiasClamp,
			bodyData1.sqrtInvInertia, bodyData1.body2World);
#endif

		//Read in 16 bytes at a time, we take 3 threads to read in a single inertia tensor, and we have some spare bandwidth. We can read
		//32 inertia tensors in 3 passes

		const PxU32 descStride2 = descStride*2;

		for (PxU32 i = 0; i < descStride2; i += 32)
		{
			PxU32 idx = i + threadIndexInWarp;
			PxU32 bodyToLoad = idx/2;

			PxU32 bodyIdx = __shfl_sync(FULL_MASK, bodyAIndex, bodyToLoad);

			if (idx < descStride2)
			{
				PxU32 offset = idx &1;
				float4* val = reinterpret_cast<float4*>(&solverTxIDatas[bodyIdx].sqrtInvInertia.column0.y);
				const PxU32 ind = (threadIdx.x / warpSize) * warpSize + bodyToLoad;
				//volatile float* sh = reinterpret_cast<volatile float*>(&inertias[threadIdx.x / 32][bodyToLoad]);
				volatile float* sh = reinterpret_cast<volatile float*>(&inertias[ind]);

				float4 v = val[offset];

				float v0 = solverTxIDatas[bodyIdx].sqrtInvInertia.column0.x;

				sh[1 + offset * 4] = v.x;
				sh[2 + offset * 4] = v.y;
				sh[3 + offset * 4] = v.z;
				sh[4 + offset * 4] = v.w;

				if(offset == 0)
					sh[offset*4] = v0;
			}
		}

		__syncwarp();

		PxMat33 invInertia0;
		const PxU32 index = (threadIdx.x / warpSize) * warpSize + threadIndexInWarp;
		if (threadIndexInWarp < descStride)
		{	
			invInertia0.column0.x = inertias[index].column0.x;
			invInertia0.column0.y = inertias[index].column0.y;
			invInertia0.column0.z = inertias[index].column0.z;
			invInertia0.column1.x = inertias[index].column1.x;
			invInertia0.column1.y = inertias[index].column1.y;
			invInertia0.column1.z = inertias[index].column1.z;
			invInertia0.column2.x = inertias[index].column2.x;
			invInertia0.column2.y = inertias[index].column2.y;
			invInertia0.column2.z = inertias[index].column2.z;

			//printf("%i: (%f, %f, %f) (%f, %f, %f) (%f, %f, %f)\n", threadIdx.x, invInertia0.column0.x, invInertia0.column0.y, invInertia0.column0.z, invInertia0.column1.x, invInertia0.column1.y, invInertia0.column1.z, invInertia0.column2.x, invInertia0.column2.y, invInertia0.column2.z);
		}

		__syncwarp(); //Required (racecheck confirmed) because inertias (Ptr sh points to inertias) is written below and read above

		for (PxU32 i = 0; i < descStride2; i += 32)
		{
			PxU32 idx = i + threadIndexInWarp;
			PxU32 bodyToLoad = idx / 2;

			PxU32 bodyIdx = __shfl_sync(FULL_MASK, bodyBIndex, bodyToLoad);

			if (idx < descStride2)
			{
				PxU32 offset = idx & 1;
				float4* val = reinterpret_cast<float4*>(&solverTxIDatas[bodyIdx].sqrtInvInertia.column0.y);
				const PxU32 ind = (threadIdx.x / warpSize) * warpSize + bodyToLoad;
				volatile float* sh = reinterpret_cast<volatile float*>(&inertias[ind]);

				float4 v = val[offset];

				float v0 = solverTxIDatas[bodyIdx].sqrtInvInertia.column0.x;

				sh[1 + offset * 4] = v.x;
				sh[2 + offset * 4] = v.y;
				sh[3 + offset * 4] = v.z;
				sh[4 + offset * 4] = v.w;

				if (offset == 0)
					sh[offset * 4] = v0;
			}
		}

		__syncwarp();

		PxMat33 invInertia1;

		if (threadIndexInWarp < descStride)
		{
			invInertia1.column0.x = inertias[index].column0.x;
			invInertia1.column0.y = inertias[index].column0.y;
			invInertia1.column0.z = inertias[index].column0.z;
			invInertia1.column1.x = inertias[index].column1.x;
			invInertia1.column1.y = inertias[index].column1.y;
			invInertia1.column1.z = inertias[index].column1.z;
			invInertia1.column2.x = inertias[index].column2.x;
			invInertia1.column2.y = inertias[index].column2.y;
			invInertia1.column2.z = inertias[index].column2.z;
		}

		//mDescStride might less than 32, we need to guard against it
		if(threadIndexInWarp < descStride)
		{
			//port contact code
			PxgBlockContactData& contactData = contactCurrentPrepPool[descIndexBatch];
			PxgBlockContactPoint* baseContact = contactBase + batch.blockContactIndex;
			PxgBlockFrictionPatch& frictionPatch = currFrictionPatches[descIndexBatch];
			PxgBlockFrictionAnchorPatch& fAnchor = currFrictionAnchors[descIndexBatch];

			//Fill in correlation information for next frame...

			PxgBlockWorkUnit& unit = workUnits[descIndexBatch];

			PxgBlockFrictionIndex index;
			index.createPatchIndex(descIndexBatch, threadIndexInWarp);

			//PxU32 frictionIndex = unit.mFrictionIndex[threadIndexInWarp];
			PxU32 edgeIndex = unit.mEdgeIndex[threadIndexInWarp];
			PxU32 frictionIndex = edgeIndex + totalCurrentEdges * unit.mPatchIndex[threadIndexInWarp];
			PxgBlockFrictionIndex* targetIndex = &frictionIndices[frictionIndex];
				
			*reinterpret_cast<uint2*>(targetIndex) = reinterpret_cast<uint2&>(index);

			//KS - todo - get some of this in shared memory/registers as quickly as possible...
			PxgSolverBodyData* bodyData0 = &solverBodyDatas[bodyAIndex];
			PxgSolverBodyData* bodyData1 = &solverBodyDatas[bodyBIndex];
			//PxgSolverTxIData* txIData0 = &solverTxIDatas[bodyAIndex];
			//PxgSolverTxIData* txIData1 = &solverTxIDatas[bodyBIndex];

			const PxAlignedTransform bodyFrame0 = bodyFrames[bodyAIndex];
			const PxAlignedTransform bodyFrame1 = bodyFrames[bodyBIndex];

			//KS - temporarily read the velocities the "slow" way so we can store the inertia-scaled velocities 
			//in velocities buffer for now. We can then switch over later when we create the new prep code for the 
			//TGS solver and leave the PGS solver as-is
#if 0
			const float4 linVel_invMass0 = velocities[bodyAIndex];
			const float4 angVelXYZ_penBiasClamp0 = velocities[bodyAIndex + totalBodies];

			const float4 linVel_invMass1 = velocities[bodyBIndex];
			const float4 angVelXYZ_penBiasClamp1 = velocities[bodyBIndex + totalBodies];
#else
			const float4 linVel_invMass0 = bodyData0->initialLinVelXYZ_invMassW;
			const float4 angVelXYZ_penBiasClamp0 = bodyData0->initialAngVelXYZ_penBiasClamp;

			const float4 linVel_invMass1 = bodyData1->initialLinVelXYZ_invMassW;
			const float4 angVelXYZ_penBiasClamp1 = bodyData1->initialAngVelXYZ_penBiasClamp;
#endif

			const PxReal solverOffsetSlop = PxMax(bodyData0->offsetSlop, bodyData1->offsetSlop);

			/*if (i >= constraintPrepDesc->numContactBatches)
			{
				if(bodyBIndex != )
			}*/

			PxU32 offset = unit.mWriteback[threadIndexInWarp];
			createFinalizeSolverContactsBlockGPU(&contactData, baseContact, frictionPatch, prevFrictionPatches, fAnchor, prevFrictionAnchors, prevFrictionIndices, *bodyData0, *bodyData1, 
				invInertia0, invInertia1, bodyFrame0, bodyFrame1, linVel_invMass0, angVelXYZ_penBiasClamp0, linVel_invMass1, angVelXYZ_penBiasClamp1,
				sharedDesc->invDtF32, sharedDesc->dt, constraintPrepDesc->bounceThresholdF32, constraintPrepDesc->frictionOffsetThreshold, constraintPrepDesc->correlationDistance,
				threadIndexInWarp, offset, &contactHeaders[descIndexBatch], &frictionHeaders[descIndexBatch], &contactPoints[batch.startConstraintIndex], 
				&frictions[batch.startFrictionIndex], totalPreviousEdges, edgeIndex, constraintPrepDesc->ccdMaxSeparation, solverOffsetSlop);

			frictionPatch.patchIndex[threadIndexInWarp] = unit.mFrictionPatchIndex[threadIndexInWarp];

			PxgBlockFrictionPatch& fpatch = frictionPatch;
			if (fpatch.anchorCount[threadIndexInWarp] >= 1)
				fpatch.anchorPoints[0][threadIndexInWarp] = PxSave3(bodyFrame0.transform(PxLoad3(fAnchor.body0Anchors[0][threadIndexInWarp])));
			if (fpatch.anchorCount[threadIndexInWarp] == 2)
				fpatch.anchorPoints[1][threadIndexInWarp] = PxSave3(bodyFrame0.transform(PxLoad3(fAnchor.body0Anchors[1][threadIndexInWarp])));
		}
	}
}
